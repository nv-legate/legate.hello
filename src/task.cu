#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "task.h"
#include <cstdio>

#define THREADS_PER_BLOCK 128

using namespace Legion;

namespace legate {
  namespace hello {

    struct HelloArgs {
    public:
      char name[64];
      int count;
    };

    __global__ void gpu_hello_world(const HelloArgs args) {
      const int cnt = blockIdx.x * blockDim.x + threadIdx.x;
      if (cnt >= args.count)
        return;
      printf("Hello %s from thread %d of block %d in GPU variant\n", args.name,
             threadIdx.x, blockIdx.x);
    }

    /*static*/ int
    HelloTask::gpu_variant(const Task *task,
                           const std::vector<PhysicalRegion> &regions, Context ctx,
                           Runtime *runtime) {
      LegateDeserializer derez(task->args, task->arglen);
      HelloArgs args;
      args.count = derez.unpack_32bit_int();
      std::string name = derez.unpack_string();
      assert((name.size() + 1) < sizeof(args.name));
      strncpy(args.name, name.c_str(), sizeof(args.name));

      // You can launch CUDA kernels inside GPU variants like normal
      const int blocks = (args.count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
      gpu_hello_world<<<blocks, THREADS_PER_BLOCK>>>(args);
      // No need to synchronize as Legion handles all this for you

      return task->is_index_space ? task->index_point[0] : 1;
    }

  } // namespace hello
} // namespace legate

